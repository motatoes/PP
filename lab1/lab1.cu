#include "hip/hip_runtime.h"
//
// include files
//

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <cutil_inline.h>


//
// kernel routine
// 

__global__ void my_first_kernel(float *x)
{
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  
  //race condition ?

  x[tid - 1] = 2;
  x[tid ] = 1;
}



//
// main code
//

int main(int argc, char **argv)
{
  float *h_x, *d_x;
  int   nblocks, nthreads, nsize, n; 

  // initialise card


  // set number of blocks, and threads per block

  nblocks  = 3000;
  nthreads = 32;
  nsize    = nblocks*nthreads ;

  // allocate memory for array

  h_x = (float *)malloc(nsize*sizeof(float));


  cutilSafeCall(hipMalloc((void **)&d_x, nsize*sizeof(float)));

  // execute kernel

  my_first_kernel<<<nblocks,nthreads>>>(d_x);
  cutilCheckMsg("my_first_kernel execution failed\n");

  hipDeviceSynchronize();


  // copy back results and print them out

  cutilSafeCall( hipMemcpy(h_x,d_x,nsize*sizeof(float),
                 hipMemcpyDeviceToHost) );

  for (n=0; n<nsize; n++) printf(" n,  x  =  %d  %f \n",n,h_x[n]);

  // free memory 

  cutilSafeCall(hipFree(d_x));
  free(h_x);

  return 0;
}

 
